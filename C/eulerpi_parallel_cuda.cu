#include "hip/hip_runtime.h"
// =================================================================
//
// File: eulerpi_parallel_cuda.cu
// Author(s):
//          Roberto Carlos Guzmán Cortés A01702388
//
// Description: This file contains the code to calculate an
//              approximation of pi using the euler series
//              algorithm. How to compile using math.h library:
//              nvcc eulerpi_parallel_cuda.cu -lm
//
// SIZE = 100_000_000
// Serial time: 1,527.14200 ms
// Parallel time: 0.00300 ms
// Speedup: 509,047.3333
//
// Copyright (c) 2020 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define SIZE 100000000 //1e8
#define THREADS	256
#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))

__global__ void calculate_pi(double *result) {
    __shared__ double cache[THREADS];

    int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	int cacheIndex = threadIdx.x;

    double acum = 0;
	while (tid < SIZE) {
		acum += pow(-1, tid)*4 / ((tid*2)+1);
		tid += blockDim.x * gridDim.x;
	}

    cache[cacheIndex] = acum;

	__syncthreads();

    int i = blockDim.x / 2;
	while (i > 0) {
		if (cacheIndex < i) {
			cache[cacheIndex] += cache[cacheIndex + i];
		}
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0) {
		result[blockIdx.x] = cache[cacheIndex];
	}
}

int main(int argc, char* argv[]){
    double *results, *d_r;
    double ms;

    results = (double*) malloc( BLOCKS * sizeof(double));
    hipMalloc( (void**) &d_r, BLOCKS * sizeof(double) );

    printf("Starting...\n");
    ms = 0;
    for(int i=0; i<N; i++){
        start_timer();
        calculate_pi<<< BLOCKS, THREADS >>>(d_r);
        ms += stop_timer();
    }

    hipMemcpy(results, d_r, BLOCKS * sizeof(long), hipMemcpyDeviceToHost);
    double acum = 0;
    for (int i = 0; i < BLOCKS; i++) {
		acum += results[i];
	}

    printf("Calculated PI = %lf\n", acum);
    printf("Real PI = %f\n", M_PI);
    printf("Percent error = %f\n", abs(acum-M_PI)/M_PI * 100);
	printf("avg time = %.5lf ms\n", (ms / N));

	hipFree(d_r);

	free(results);
    return 0;
}